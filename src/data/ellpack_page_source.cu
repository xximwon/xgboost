#include "hip/hip_runtime.h"
/**
 * Copyright 2019-2024, XGBoost contributors
 */
#include <sys/sysinfo.h>         // for sysinfo

#include <cstddef>  // for size_t
#include <cstdint>  // for int8_t, uint64_t, uint32_t
#include <memory>   // for shared_ptr, make_unique, make_shared
#include <numeric>  // for accumulate
#include <utility>  // for move

#include "../common/common.h"               // for safe_cuda
#include "../common/common.h"               // for HumanMemUnit
#include "../common/cuda_rt_utils.h"        // for SetDevice
#include "../common/device_helpers.cuh"     // for CUDAStreamView, DefaultStream
#include "../common/ref_resource_view.cuh"  // for MakeFixedVecWithCudaMalloc
#include "../common/resource.cuh"           // for PrivateCudaMmapConstStream
#include "../common/transform_iterator.h"   // for MakeIndexTransformIter
#include "ellpack_page.cuh"                 // for EllpackPageImpl
#include "ellpack_page.h"                   // for EllpackPage
#include "ellpack_page_source.h"
#include "proxy_dmatrix.cuh"            // for Dispatch
#include "xgboost/base.h"               // for bst_idx_t
#include "xgboost/collective/socket.h"  // fixme

namespace xgboost::data {

#if !defined(xgboost_CHECK_SYS_CALL)
#define xgboost_CHECK_SYS_CALL(exp, expected)         \
  do {                                                \
    if (XGBOOST_EXPECT((exp) != (expected), false)) { \
      ::xgboost::system::ThrowAtError(#exp);          \
    }                                                 \
  } while (false)
#endif  // !defined(xgboost_CHECK_SYS_CALL)

[[nodiscard]] std::size_t AvailableHostMemory() {
  struct sysinfo info;
  xgboost_CHECK_SYS_CALL(sysinfo(&info), 0);
  return info.freeram;
}

/**
 * Cache
 */
EllpackHostCache::EllpackHostCache(bst_idx_t n_batches, bool prefer_device,
                                   std::vector<std::size_t> cache_mapping,
                                   std::vector<std::size_t> buffer_bytes,
                                   std::vector<std::size_t> base_rows,
                                   std::vector<bst_idx_t> buffer_rows)
    : total_available_mem{dh::TotalMemory(curt::CurrentDevice())},
      n_batches_orig{n_batches},
      prefer_device{prefer_device},
      cache_mapping{std::move(cache_mapping)},
      buffer_bytes{std::move(buffer_bytes)},
      base_rows{std::move(base_rows)},
      buffer_rows{std::move(buffer_rows)} {};

EllpackHostCache::~EllpackHostCache() = default;

[[nodiscard]] std::size_t EllpackHostCache::SizeBytes() const {
  auto it = common::MakeIndexTransformIter([&](auto i) { return pages.at(i)->MemCostBytes(); });
  return std::accumulate(it, it + pages.size(), 0ul);
}

EllpackPageImpl const* EllpackHostCache::At(std::int32_t k) {
  return this->pages.at(k).get();
}

/**
 * Cache stream.
 */
class EllpackHostCacheStreamImpl {
  std::shared_ptr<EllpackHostCache> cache_;
  std::int32_t ptr_{0};

 public:
  explicit EllpackHostCacheStreamImpl(std::shared_ptr<EllpackHostCache> cache)
      : cache_{std::move(cache)} {}

  auto Share() { return cache_; }

  void Seek(bst_idx_t offset_bytes) {
    std::size_t n_bytes{0};
    std::int32_t k{-1};
    for (std::size_t i = 0, n = cache_->pages.size(); i < n; ++i) {
      if (n_bytes == offset_bytes) {
        k = i;
        break;
      }
      n_bytes += cache_->pages[i]->MemCostBytes();
    }
    if (offset_bytes == n_bytes && k == -1) {
      k = this->cache_->pages.size();  // seek end
    }
    CHECK_NE(k, -1) << "Invalid offset:" << offset_bytes;
    ptr_ = k;
  }

  [[nodiscard]] bool Write(EllpackPage const& page) {
    auto impl = page.Impl();

    auto ctx = Context{}.MakeCUDA(dh::CurrentDevice());

    this->cache_->sizes_orig.push_back(page.Impl()->MemCostBytes());
    auto orig_ptr = this->cache_->sizes_orig.size() - 1;

    auto cache_idx = this->cache_->cache_mapping.at(orig_ptr);
    // Wrap up the previous page if this is a new page, or this is the last page.
    auto new_page = cache_idx == this->cache_->pages.size();
    auto last_page = (orig_ptr + 1) == this->cache_->n_batches_orig;

    auto commit_page = [this, &ctx] {
      auto const& d_page = this->cache_->pages.back();

      auto& old_impl = this->cache_->pages.back();
      CHECK_EQ(old_impl->gidx_buffer.Resource()->Type(), common::ResourceHandler::kCudaMalloc);

      auto new_impl = std::make_unique<EllpackPageImpl>();
      new_impl->CopyInfo(d_page.get());
      new_impl->gidx_buffer = common::MakeFixedVecWithPinnedMalloc<common::CompressedByteT>(
          old_impl->gidx_buffer.size());
      dh::safe_cuda(hipMemcpyAsync(new_impl->gidx_buffer.data(), old_impl->gidx_buffer.data(),
                                    old_impl->gidx_buffer.size_bytes(), hipMemcpyDefault));

      old_impl.reset();
      this->cache_->pages.back() = std::move(new_impl);
      LOG(INFO) << "Create cache page with size:"
                << common::HumanMemUnit(this->cache_->pages.back()->MemCostBytes());
    };

    if (new_page) {
      if (!this->cache_->pages.empty()) {
        // New to wrap up the previous page.
        commit_page();
      }
      // Push a new page
      auto n_bytes = this->cache_->buffer_bytes.at(this->cache_->pages.size());
      auto n_samples = this->cache_->buffer_rows.at(this->cache_->pages.size());
      auto new_impl = std::make_unique<EllpackPageImpl>(&ctx, impl->CutsShared(), impl->IsDense(),
                                                        impl->info.row_stride, n_samples);
      new_impl->base_rowid = this->cache_->base_rows.at(orig_ptr);
      CHECK_EQ(this->cache_->base_rows.at(orig_ptr), impl->base_rowid);
      new_impl->SetNumSymbols(impl->NumSymbols());
      new_impl->gidx_buffer = common::MakeFixedVecWithCudaMalloc<common::CompressedByteT>(n_bytes);
      dh::safe_cuda(hipMemsetAsync(new_impl->gidx_buffer.data(), '\0',
                                    new_impl->gidx_buffer.size_bytes(), ctx.CUDACtx()->Stream()));
      auto offset = new_impl->Copy(&ctx, impl, 0);

      this->cache_->offsets.push_back(offset);
      this->cache_->pages.push_back(std::move(new_impl));
    } else {
      CHECK(!this->cache_->pages.empty());
      CHECK_EQ(cache_idx, this->cache_->pages.size() - 1);
      auto& new_impl = this->cache_->pages.back();
      auto offset = new_impl->Copy(&ctx, impl, this->cache_->offsets.back());
      this->cache_->offsets.back() += offset;
      if (last_page) {
        commit_page();
      }
    }

    return new_page;
  }

  void Read(EllpackPage* out, bool prefetch_copy) const {
    auto page = this->cache_->At(ptr_);

    auto impl = out->Impl();
    if (prefetch_copy) {
      impl->gidx_buffer =
          common::MakeFixedVecWithCudaMalloc<common::CompressedByteT>(page->gidx_buffer.size());
      dh::safe_cuda(hipMemcpyAsync(impl->gidx_buffer.data(), page->gidx_buffer.data(),
                                    page->gidx_buffer.size_bytes(), hipMemcpyDefault));
    } else {
      auto res = page->gidx_buffer.Resource();
      impl->gidx_buffer = common::RefResourceView<common::CompressedByteT>{
          res->DataAs<common::CompressedByteT>(), page->gidx_buffer.size(), res};
    }

    impl->CopyInfo(page);
  }
};

/**
 * EllpackHostCacheStream
 */
EllpackHostCacheStream::EllpackHostCacheStream(std::shared_ptr<EllpackHostCache> cache)
    : p_impl_{std::make_unique<EllpackHostCacheStreamImpl>(std::move(cache))} {}

EllpackHostCacheStream::~EllpackHostCacheStream() = default;

std::shared_ptr<EllpackHostCache> EllpackHostCacheStream::Share() { return p_impl_->Share(); }

void EllpackHostCacheStream::Seek(bst_idx_t offset_bytes) { this->p_impl_->Seek(offset_bytes); }

void EllpackHostCacheStream::Read(EllpackPage* page, bool prefetch_copy) const {
  this->p_impl_->Read(page, prefetch_copy);
}

[[nodiscard]] bool EllpackHostCacheStream::Write(EllpackPage const& page) {
  return this->p_impl_->Write(page);
}

/**
 * EllpackCacheStreamPolicy
 */
template <typename S, template <typename> typename F>
[[nodiscard]] std::unique_ptr<typename EllpackCacheStreamPolicy<S, F>::WriterT>
EllpackCacheStreamPolicy<S, F>::CreateWriter(StringView, std::uint32_t iter) {
  if (!this->p_cache_) {
    this->p_cache_ = std::make_shared<EllpackHostCache>(this->OrigBatches(), this->PreferDevice(),
                                                        this->CacheMapping(), this->BufferBytes(),
                                                        this->BaseRows(), this->BufferRows());
  }
  auto fo = std::make_unique<EllpackHostCacheStream>(this->p_cache_);
  if (iter == 0) {
    CHECK(this->p_cache_->Empty());
  } else {
    fo->Seek(this->p_cache_->SizeBytes());
  }
  return fo;
}

template <typename S, template <typename> typename F>
[[nodiscard]] std::unique_ptr<typename EllpackCacheStreamPolicy<S, F>::ReaderT>
EllpackCacheStreamPolicy<S, F>::CreateReader(StringView, bst_idx_t offset, bst_idx_t) const {
  auto fi = std::make_unique<ReaderT>(this->p_cache_);
  fi->Seek(offset);
  return fi;
}

// Instantiation
template std::unique_ptr<
    typename EllpackCacheStreamPolicy<EllpackPage, EllpackFormatPolicy>::WriterT>
EllpackCacheStreamPolicy<EllpackPage, EllpackFormatPolicy>::CreateWriter(StringView name,
                                                                         std::uint32_t iter);

template std::unique_ptr<
    typename EllpackCacheStreamPolicy<EllpackPage, EllpackFormatPolicy>::ReaderT>
EllpackCacheStreamPolicy<EllpackPage, EllpackFormatPolicy>::CreateReader(StringView name,
                                                                         bst_idx_t offset,
                                                                         bst_idx_t length) const;

/**
 * EllpackMmapStreamPolicy
 */

template <typename S, template <typename> typename F>
[[nodiscard]] std::unique_ptr<typename EllpackMmapStreamPolicy<S, F>::ReaderT>
EllpackMmapStreamPolicy<S, F>::CreateReader(StringView name, bst_idx_t offset,
                                            bst_idx_t length) const {
  if (has_hmm_) {
    return std::make_unique<common::PrivateCudaMmapConstStream>(name, offset, length);
  } else {
    return std::make_unique<common::PrivateMmapConstStream>(name, offset, length);
  }
}

// Instantiation
template std::unique_ptr<
    typename EllpackMmapStreamPolicy<EllpackPage, EllpackFormatPolicy>::ReaderT>
EllpackMmapStreamPolicy<EllpackPage, EllpackFormatPolicy>::CreateReader(StringView name,
                                                                        bst_idx_t offset,
                                                                        bst_idx_t length) const;

void CalcCachInfo(Context const* ctx, bool is_dense,
                  std::shared_ptr<common::HistogramCuts const> cuts, double min_page_bytes,
                  ExternalDataInfo const& ext_info) {
  auto ell_info = CalcNumSymbols(ctx, ext_info.row_stride, is_dense, cuts);
  std::vector<std::size_t> cache_size;
  CHECK_EQ(ext_info.base_rows.size(), ext_info.n_batches + 1);
  std::vector<std::size_t> cache_mapping(ext_info.base_rows.size(), 0);
  std::vector<std::size_t> cache_rows;

  for (std::size_t i = 0; i < ext_info.n_batches; ++i) {
    auto n_samples = ext_info.base_rows.at(i + 1) - ext_info.base_rows[i];
    auto n_bytes = common::CompressedBufferWriter::CalculateBufferSize(
        ext_info.row_stride * n_samples, ell_info.n_symbols);
    if (cache_size.empty()) {
      cache_size.push_back(n_bytes);
      cache_rows.push_back(n_samples);
    } else if (cache_size.back() < min_page_bytes) {
      cache_size.back() += n_bytes;
      cache_rows.back() += n_samples;
    } else {
      cache_size.push_back(n_bytes);
      cache_rows.push_back(n_samples);
    }
    cache_mapping[i] = cache_size.size() - 1;
  }
}

/**
 * EllpackPageSourceImpl
 */
template <typename F>
void EllpackPageSourceImpl<F>::Fetch() {
  curt::SetDevice(this->Device().ordinal);
  if (!this->ReadCache()) {
    if (this->Iter() != 0 && !this->sync_) {
      // source is initialized to be the 0th page during construction, so when count_ is 0
      // there's no need to increment the source.
      ++(*this->source_);
    }
    // This is not read from cache so we still need it to be synced with sparse page source.
    CHECK_EQ(this->Iter(), this->source_->Iter());
    auto const& csr = this->source_->Page();
    this->page_.reset(new EllpackPage{});
    auto* impl = this->page_->Impl();
    Context ctx = Context{}.MakeCUDA(this->Device().ordinal);
    *impl = EllpackPageImpl{&ctx, this->GetCuts(), *csr, is_dense_, row_stride_, feature_types_};
    this->page_->SetBaseRowId(csr->base_rowid);
    LOG(INFO) << "Generated an Ellpack page with size: "
              << common::HumanMemUnit(impl->MemCostBytes())
              << " from a SparsePage with size:" << common::HumanMemUnit(csr->MemCostBytes());
    this->WriteCache();
  }
}

// Instantiation
template void
EllpackPageSourceImpl<DefaultFormatStreamPolicy<EllpackPage, EllpackFormatPolicy>>::Fetch();
template void
EllpackPageSourceImpl<EllpackCacheStreamPolicy<EllpackPage, EllpackFormatPolicy>>::Fetch();
template void
EllpackPageSourceImpl<EllpackMmapStreamPolicy<EllpackPage, EllpackFormatPolicy>>::Fetch();

/**
 * ExtEllpackPageSourceImpl
 */
template <typename F>
void ExtEllpackPageSourceImpl<F>::Fetch() {
  curt::SetDevice(this->Device().ordinal);
  if (!this->ReadCache()) {
    auto iter = this->source_->Iter();
    CHECK_EQ(this->Iter(), iter);
    cuda_impl::Dispatch(proxy_, [this](auto const& value) {
      CHECK(this->proxy_->Ctx()->IsCUDA()) << "All batches must use the same device type.";
      proxy_->Info().feature_types.SetDevice(dh::GetDevice(this->ctx_));
      auto d_feature_types = proxy_->Info().feature_types.ConstDeviceSpan();
      auto n_samples = value.NumRows();

      dh::device_vector<size_t> row_counts(n_samples + 1, 0);
      common::Span<size_t> row_counts_span(row_counts.data().get(), row_counts.size());
      GetRowCounts(this->ctx_, value, row_counts_span, dh::GetDevice(this->ctx_), this->missing_);
      this->page_.reset(new EllpackPage{});
      *this->page_->Impl() = EllpackPageImpl{this->ctx_,
                                             value,
                                             this->missing_,
                                             this->info_->IsDense(),
                                             row_counts_span,
                                             d_feature_types,
                                             this->ext_info_.row_stride,
                                             n_samples,
                                             this->GetCuts()};
      this->info_->Extend(proxy_->Info(), false, true);
    });
    // The size of ellpack is logged in write cache.
    LOG(INFO) << "Estimated batch size:"
              << cuda_impl::Dispatch<false>(proxy_, [](auto const& adapter) {
                   return common::HumanMemUnit(adapter->SizeBytes());
                 });
    this->page_->SetBaseRowId(this->ext_info_.base_rows.at(iter));
    this->WriteCache();
  }
}

// Instantiation
template void
ExtEllpackPageSourceImpl<DefaultFormatStreamPolicy<EllpackPage, EllpackFormatPolicy>>::Fetch();
template void
ExtEllpackPageSourceImpl<EllpackCacheStreamPolicy<EllpackPage, EllpackFormatPolicy>>::Fetch();
template void
ExtEllpackPageSourceImpl<EllpackMmapStreamPolicy<EllpackPage, EllpackFormatPolicy>>::Fetch();
}  // namespace xgboost::data
